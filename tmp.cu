#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
	printf("Hello World du GPU\n");
}

int main(void) {
	printf("Hello World du CPU\n");
	cuda_hello<<<1,1>>>();
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize error: %s\n", hipGetErrorString(cudaStatus));
		// Additional error handling if needed
	}
	return 0;
}